
#include <nppi.h>
#include "cuda_kernels.h"

int GetCUDADeviceCount()
{
	int count;
	hipError_t error = hipGetDeviceCount(&count);

	if (error == hipSuccess)
	{
		return count;
	}
	else
	{
		return 0;
	}
}

Npp8u * device_nv12[2] = { NULL, NULL };
Npp8u * device_yuv420[3] = { NULL, NULL, NULL };
Npp8u * device_data = NULL;
Npp8u * device_data_resized = NULL;

void init_cuda_memory(int w, int h, int W, int H)
{
	checkCuda(hipMalloc(&device_nv12[0], W * H * sizeof(Npp8u)));
	checkCuda(hipMalloc(&device_nv12[1], ((W * H) / 2) * sizeof(Npp8u)));

	checkCuda(hipMalloc(&device_yuv420[0], (W * H) * sizeof(Npp8u)));
	checkCuda(hipMalloc(&device_yuv420[1], ((W * H)/4) * sizeof(Npp8u)));
	checkCuda(hipMalloc(&device_yuv420[2], ((W * H)/4) * sizeof(Npp8u)));

	checkCuda(hipMalloc(&device_data, ((W * H) * 4) * sizeof(Npp8u)));

	if (w != W)
	{
		checkCuda(hipMalloc(&device_data_resized, ((w * h) * 4) * sizeof(Npp8u)));
	}
	else
	{
		device_data_resized = NULL;
	}
}

void release_cuda_memory()
{
	checkCuda(hipFree(device_nv12[0]));
	checkCuda(hipFree(device_nv12[1]));
	
	checkCuda(hipFree(device_yuv420[0]));
	checkCuda(hipFree(device_yuv420[1]));
	checkCuda(hipFree(device_yuv420[2]));

	checkCuda(hipFree(device_data));

	if (device_data_resized != NULL)
	{
		checkCuda(hipFree(device_data_resized));
	}
}

int NV12_to_BGRA(unsigned char *src_y, unsigned char *src_uv, int src_linesize, unsigned char *dst_data, int w, int h, int W, int H)
{
	NppStatus err;	
	int nSrcPitchCUDA, res = 0;

	checkCuda(hipMemcpy(device_nv12[0], src_y,
		W * H * sizeof(Npp8u), hipMemcpyHostToDevice));
	checkCuda(hipMemcpy(device_nv12[1], src_uv,
		((W * H) / 2) * sizeof(Npp8u), hipMemcpyHostToDevice));

	int aDstStep[3] = { W, W / 2, W / 2 };
	err = nppiNV12ToYUV420_8u_P2P3R(device_nv12, W, device_yuv420, aDstStep, NppiSize{ W, H });

	if (err == NPP_SUCCESS) {
		err = nppiYUV420ToBGR_8u_P3C4R(device_yuv420, aDstStep, device_data, (W * 4), NppiSize{ W, H });
	}

	if (err == NPP_SUCCESS) {
		if (w != W) {
			err = nppiResize_8u_C4R(device_data, (W * 4), NppiSize{ W, H }, NppiRect{ 0, 0,  W, H }, device_data_resized, (w * 4), NppiSize{ w, h }, NppiRect{ 0, 0,  w, h }, NPPI_INTER_LINEAR);
		}
	}

	if (err == NPP_SUCCESS){
		if (w != W) {
			checkCuda(hipMemcpy(dst_data, device_data_resized,
				((w * h) * 4) * sizeof(Npp8u), hipMemcpyDeviceToHost));
		}
		else
		{
			checkCuda(hipMemcpy(dst_data, device_data,
				((w * h) * 4) * sizeof(Npp8u), hipMemcpyDeviceToHost));
		}

		res = 1;
	}	
	
	return res;
}
