
#include <nppi.h>
#include "cuda_kernels.h"

int GetCUDADeviceCount()
{
	int count;
	hipError_t error = hipGetDeviceCount(&count);

	if (error == hipSuccess)
	{
		return count;
	}
	else
	{
		return 0;
	}
}

int NV21_to_BGRA(unsigned char *src_y, unsigned char *src_uv, int src_linesize,
	unsigned char *dst_data, int w, int h)
{
	NppStatus err;
	Npp8u * pSrc[2];
	int nSrcPitchCUDA;
	pSrc[0] = src_y; // nppiMalloc_8u_C1(w, h, &nSrcPitchCUDA);
	pSrc[1] = src_uv;
	/*
	Npp8u *pSrcImageCUDA 
	NPP_ASSERT_NOT_NULL(pSrcImageCUDA);
	// copy image loaded via FreeImage to into CUDA device memory, i.e.
	// transfer the image-data up to the GPU's video-memory
	NPP_CHECK_CUDA(hipMemcpy2D(pSrcImageCUDA, nSrcPitchCUDA, pSrcData, nSrcPitch,
		nImageWidth, nImageHeight, hipMemcpyHostToDevice));
		*/
	err = nppiNV21ToBGR_8u_P2C4R(pSrc, src_linesize,
								dst_data, w*4,
								NppiSize{w, h});
	if (err != NPP_SUCCESS){		    
		return -1;
	}
	
	return 0;
}
