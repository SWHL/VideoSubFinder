
#include <nppi.h>
#include "cuda_kernels.h"

int GetCUDADeviceCount()
{
	int count;
	hipError_t error = hipGetDeviceCount(&count);

	if (error == hipSuccess)
	{
		return count;
	}
	else
	{
		return 0;
	}
}

int NV12_to_BGR(unsigned char *src_y, unsigned char *src_uv, int src_linesize, unsigned char *dst_data, int w, int h, int W, int H)
{
	NppStatus err;	
	int nSrcPitchCUDA, res = 0;

	Npp8u* device_nv12[2] = { NULL, NULL };
	Npp8u* device_BGR = NULL;
	Npp8u* device_BGR_resized = NULL;

	checkCuda(hipMalloc(&device_nv12[0], W * H * sizeof(Npp8u)));
	checkCuda(hipMalloc(&device_nv12[1], ((W * H) / 2) * sizeof(Npp8u)));

	checkCuda(hipMalloc(&device_BGR, W * H * 3 * sizeof(Npp8u)));

	if (w != W)
	{
		checkCuda(hipMalloc(&device_BGR_resized, ((w * h) * 3) * sizeof(Npp8u)));
	}

	checkCuda(hipMemcpy(device_nv12[0], src_y,
		W * H * sizeof(Npp8u), hipMemcpyHostToDevice));
	checkCuda(hipMemcpy(device_nv12[1], src_uv,
		((W * H) / 2) * sizeof(Npp8u), hipMemcpyHostToDevice));


	//err = nppiNV12ToBGR_8u_P2C3R(device_nv12, W, device_BGR, (W * 3), NppiSize{ W, H });
	//err = nppiNV12ToBGR_709HDTV_8u_P2C3R(device_nv12, W, device_BGR, (W * 3), NppiSize{ W, H });	
	err = nppiNV12ToBGR_709CSC_8u_P2C3R(device_nv12, W, device_BGR, (W * 3), NppiSize{ W, H });

	if (err == NPP_SUCCESS) {
		if (w != W) {
			err = nppiResize_8u_C3R(device_BGR, (W * 3), NppiSize{ W, H }, NppiRect{ 0, 0,  W, H }, device_BGR_resized, (w * 3), NppiSize{ w, h }, NppiRect{ 0, 0,  w, h }, NPPI_INTER_LINEAR);
		}
	}

	if (err == NPP_SUCCESS){
		if (w != W) {
			checkCuda(hipMemcpy(dst_data, device_BGR_resized,
				((w * h) * 3) * sizeof(Npp8u), hipMemcpyDeviceToHost));
		}
		else
		{
			checkCuda(hipMemcpy(dst_data, device_BGR,
				((w * h) * 3) * sizeof(Npp8u), hipMemcpyDeviceToHost));
		}

		res = 1;
	}

	checkCuda(hipFree(device_nv12[0]));
	checkCuda(hipFree(device_nv12[1]));
	checkCuda(hipFree(device_BGR));
	if (device_BGR_resized)
	{
		checkCuda(hipFree(device_BGR_resized));
	}
	
	return res;
}
