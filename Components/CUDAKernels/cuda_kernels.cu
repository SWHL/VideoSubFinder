
#include "cuda_kernels.h"

int GetCUDADeviceCount()
{
	int count;
	hipError_t error = hipGetDeviceCount(&count);

	if (error == hipSuccess)
	{
		return count;
	}
	else
	{
		return 0;
	}
}
